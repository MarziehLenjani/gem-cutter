#include "hip/hip_runtime.h"
/*
 * PROJECT: Thread-cooperative FM-index on GPU
 * FILE: genIndex.h
 * DATE: 1/9/2015
 * AUTHOR(S): Alejandro Chacon <alejandro.chacon@uab.es>
 * DESCRIPTION: FM-Index DNA backward-search customized for GEM Mapper
 */

#include "../include/gpu_fmi_core.h"


void __global__ gpu_fmi_search_kernel(const gpu_fmi_device_entry_t *fmi, const uint64_t bwtSize,
									  const uint32_t numSeeds, const ulonglong2 *seeds, ulonglong2 *resIntervals)
{
	const uint32_t globalThreadIdx     = blockIdx.x * GPU_MAX_THREADS_PER_SM + threadIdx.x;
	const uint32_t localWarpThreadIdx  = globalThreadIdx     % GPU_WARP_SIZE;
	const uint32_t localEntryIdx       = localWarpThreadIdx  / GPU_FMI_THREADS_PER_ENTRY;
	const uint32_t localEntryThreadIdx = localWarpThreadIdx  % GPU_FMI_THREADS_PER_ENTRY;
	const uint32_t idSeed 	 		   = globalThreadIdx     / GPU_FMI_SEED_THREADS_PER_ENTRY;

	if ( (threadIdx.x < GPU_MAX_THREADS_PER_SM) && (globalThreadIdx < (numSeeds * GPU_FMI_SEED_THREADS_PER_ENTRY)) ){

		const uint32_t   localIdSeed = idSeed % GPU_FMI_SEED_ENTRIES_PER_WARP;
		const ulonglong2 seed 	     = seeds[idSeed];
		const uint32_t   seedSize    = seed.y >> (GPU_UINT64_LENGTH - GPU_FMI_SEED_FIELD_SIZE);
			  uint64_t   currentSeed = seed.x;

			  uint64_t sharedInterval, interval = (localEntryIdx % GPU_FMI_ENTRIES_PER_SEED) ? bwtSize : 0;
			  uint32_t idStep = 0, foundSeed = 0;

		__shared__ gpu_fmi_exch_bmp_mem_t   exchBMP[GPU_FMI_ENTRIES_PER_BLOCK];
				   gpu_fmi_exch_bmp_mem_t * seedExchBMP = &exchBMP[threadIdx.x / GPU_FMI_THREADS_PER_ENTRY];

		while((idStep < seedSize) && (foundSeed == 0)){			
			const uint64_t entryIdx    	  =  interval / GPU_FMI_ENTRY_SIZE;
			const uint32_t bitmapPosition =  interval % GPU_FMI_ENTRY_SIZE;

			// Gathering the base of the seed
			currentSeed = (idStep == GPU_FMI_SEED_BASES_PER_ENTRY) ? seed.y : currentSeed;
			const uint32_t bit0 =  currentSeed & 0x1L;
			const uint32_t bit1 = (currentSeed & 0x2L) >> 1;
			currentSeed >>= GPU_FMI_SEED_CHAR_LENGTH;
 
			// Loading FM-index entry in thread cooperative way
			const uint32_t missedEntry   = (entryIdx % GPU_FMI_ALTERNATE_COUNTERS != bit1) ? 1 : 0;
			const uint64_t entryIdxFixed = (localEntryThreadIdx == 0) ? entryIdx + missedEntry : entryIdx;
			uint4 loadEntry              = fmi[entryIdxFixed].v[localEntryThreadIdx];

			// Compute LF-Mapping (th0 of each group contain the result)
 			interval = LF_Mapping(loadEntry, seedExchBMP, missedEntry, localEntryThreadIdx, bitmapPosition, bit1, bit0);

			// Shared results
			const uint32_t lane = (localEntryThreadIdx == 0) ? localWarpThreadIdx + GPU_FMI_THREADS_PER_ENTRY : localEntryIdx * GPU_FMI_THREADS_PER_ENTRY;
			sharedInterval      = shfl_64(interval, lane);

			// Early exit condition
			foundSeed           = (__ballot(interval == sharedInterval) >> (localIdSeed * GPU_FMI_SEED_THREADS_PER_ENTRY)) & GPU_UINT32_MASK_ONE_LOW;

			// Update interval for bitmap threads
			if (localEntryThreadIdx) interval = sharedInterval;

			// Increment for the next Backward-Search
			idStep++;
		}
		// Save intervals
		if((localWarpThreadIdx % GPU_FMI_SEED_THREADS_PER_ENTRY) == 0)
			resIntervals[idSeed] = make_ulonglong2(interval, sharedInterval);
	}
}

extern "C"
gpu_error_t gpu_fmi_search_launch_kernel(const gpu_fmi_device_entry_t *d_fmi, const uint64_t bwtSize, const uint32_t numSeeds, const ulonglong2 *d_seeds, ulonglong2 *d_intervals)
{
	const uint32_t threads = 128;
	const uint32_t blocks  = GPU_DIV_CEIL(numSeeds * GPU_FMI_SEED_THREADS_PER_ENTRY, threads);
	const uint32_t nreps   = 10;

	float elapsed_time_ms = 0.0f;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

		for(uint32_t iteration = 0; iteration < nreps; ++iteration)
			gpu_fmi_search_kernel<<<blocks,threads>>>(d_fmi, bwtSize, numSeeds, d_seeds, d_intervals);

	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventElapsedTime(&elapsed_time_ms, start, stop);
	elapsed_time_ms /= nreps;

	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("\t Time Kernel GPU:  %8.2f ms\n", elapsed_time_ms);

	return(SUCCESS);
}

extern "C"
gpu_error_t gpu_fmi_search_process_buffer(gpu_buffer_t *mBuff)
{
	gpu_index_buffer_t 			  	 *index    	  =  mBuff->index;
	gpu_fmi_search_seeds_buffer_t 	 *seeds    	  = &mBuff->data.search.seeds;
	gpu_fmi_search_sa_inter_buffer_t *saIntervals = &mBuff->data.search.saIntervals;
	uint32_t 					     numSeeds	  =  mBuff->data.search.seeds.numSeeds;
	hipStream_t 				     idStream	  =  mBuff->idStream;
	uint32_t					     idSupDev	  =  mBuff->idSupportedDevice;

	uint32_t threadsPerBlock = GPU_MAX_THREADS_PER_BLOCK;
	uint32_t numThreads = numSeeds * GPU_FMI_SEED_THREADS_PER_ENTRY;
	uint32_t blocksPerGrid = GPU_DIV_CEIL(numThreads, threadsPerBlock);

	//printf("KEPLER 2ndGen: LAUNCH KERNEL -- Bloques: %d - Th_block %d\n", blocksPerGrid, threadsPerBlock);
	gpu_fmi_search_kernel<<<blocksPerGrid, threadsPerBlock, 0, idStream>>>((gpu_fmi_device_entry_t*) index->d_fmi[idSupDev], index->bwtSize,
																		   seeds->numSeeds, (ulonglong2*) seeds->d_seeds,
																		   (ulonglong2*) saIntervals->d_intervals);
	return(SUCCESS);
}



