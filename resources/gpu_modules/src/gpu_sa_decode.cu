#include "hip/hip_runtime.h"
/*
 *  GEM-Cutter "Highly optimized genomic resources for GPUs"
 *  Copyright (c) 2013-2016 by Alejandro Chacon    <alejandro.chacond@gmail.com>
 *
 *  Licensed under GNU General Public License 3.0 or later.
 *  Some rights reserved. See LICENSE, AUTHORS.
 *  @license GPL-3.0+ <http://www.gnu.org/licenses/gpl-3.0.en.html>
 */

#ifndef GPU_SA_DECODE_C_
#define GPU_SA_DECODE_C_

#include "../include/gpu_sa_core.h"

void __global__ gpu_sa_decoding_kernel(const uint64_t* const d_SA, const uint32_t samplingRate,
                                       const uint32_t numDecodings, const ulonglong2* const d_endBWTPos,
                                       uint64_t* const d_textPos)
{
  const uint32_t idDecoding = gpu_get_thread_idx();

  if(idDecoding < numDecodings){
    const ulonglong2 saPosition = d_endBWTPos[idDecoding];
    if((saPosition.x < GPU_UINT64_MAX_VALUE) && (saPosition.x < GPU_UINT64_MAX_VALUE))
      d_textPos[idDecoding] = d_SA[saPosition.x / samplingRate] + saPosition.y;
  }
}

extern "C"
gpu_error_t gpu_sa_decoding_launch_kernel(const uint64_t* const d_SA, const uint32_t samplingRate,
		                                  const uint32_t numDecodings, const ulonglong2* const d_endBWTPos,
		                                  uint64_t* const d_textPos)
{
  const uint32_t threads = 128;
  const uint32_t blocks  = GPU_DIV_CEIL(numDecodings, threads);
  const uint32_t nreps   = 10;

  float elapsed_time_ms = 0.0f;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

    for(uint32_t iteration = 0; iteration < nreps; ++iteration)
      gpu_sa_decoding_kernel<<<blocks,threads>>>(d_SA, samplingRate, numDecodings, d_endBWTPos, d_textPos);

  hipEventRecord(stop, 0);
  hipDeviceSynchronize();
  hipEventElapsedTime(&elapsed_time_ms, start, stop);
  elapsed_time_ms /= nreps;

  hipEventDestroy(start);
  hipEventDestroy(stop);
  printf("\t Time Kernel GPU:  %8.2f ms\n", elapsed_time_ms);

  return(SUCCESS);
}

extern "C"
gpu_error_t gpu_sa_decode_process_buffer(gpu_buffer_t* const mBuff)
{
  const gpu_index_buffer_t* const               index        =  mBuff->index;
  const gpu_fmi_decode_end_pos_buffer_t* const  endPos       = &mBuff->data.decode.endPositions;
  const gpu_fmi_decode_text_pos_buffer_t* const textPos      = &mBuff->data.decode.textPositions;
  const uint32_t                                numDecodings =  mBuff->data.decode.endPositions.numDecodings;
  const uint32_t                                samplingRate =  mBuff->data.decode.samplingRate;
  const hipStream_t                            idStream     =  mBuff->idStream;
  const uint32_t                                idSupDev     =  mBuff->idSupportedDevice;
  const gpu_device_info_t* const                device       =  mBuff->device[idSupDev];

  dim3 blocksPerGrid, threadsPerBlock;
  const uint32_t numThreads = numDecodings;
  gpu_device_kernel_thread_configuration(device, numThreads, &blocksPerGrid, &threadsPerBlock);

  gpu_sa_decoding_kernel<<<blocksPerGrid, threadsPerBlock, 0, idStream>>>(index->sa.d_sa[idSupDev], samplingRate, numDecodings,
		                                                                  (ulonglong2*) endPos->d_endBWTPos, (uint64_t*) textPos->d_textPos);

  return(SUCCESS);
}

#endif /* GPU_SA_DECODE_C_ */



