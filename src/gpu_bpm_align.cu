#include "hip/hip_runtime.h"
/*
 *  GEM-Cutter "Highly optimized genomic resources for GPUs"
 *  Copyright (c) 2011-2018 by Alejandro Chacon    <alejandro.chacond@gmail.com>
 *
 *  Licensed under GNU General Public License 3.0 or later.
 *  Some rights reserved. See LICENSE, AUTHORS.
 *  @license GPL-3.0+ <http://www.gnu.org/licenses/gpl-3.0.en.html>
 */

#ifndef GPU_BPM_ALIGN_CU_
#define GPU_BPM_ALIGN_CU_

#include "../include/gpu_bpm_core.h"
#include "../include/gpu_text_core.h"
#include "../include/gpu_scheduler_core.h"

#define GPU_BMP_ALIGN_CIGAR_LUT_OFFSET      8
#define GPU_BMP_ALIGN_CIGAR_LUT_SIZE        16
#define GPU_BMP_ALIGN_BASE_CANDIDATE_LENGTH 8
#define GPU_BMP_ALIGN_BASE_QUERY_LENGTH     8


GPU_INLINE __device__ void gpu_bpm_align_backtrace(const uint32_t* const dpPV, const uint32_t* const dpMV, const uint64_t* const query,
												   const uint64_t* const referencePlain, const uint64_t* const referenceMasked, const uint64_t sizeReference, const uint64_t posCandidate,
												   gpu_bpm_align_cigar_entry_t* const dpCIGAR, const bool leftGapAlign, const uint32_t minColumn, const uint32_t sizeQuery,
												   const uint32_t intraQueryThreadIdx, const uint32_t threadsPerQuery,
												   gpu_bpm_align_coord_t* const initCoodRes, uint32_t* const cigarLenghtRes)
{
  // Initializing back-trace threading variables
  const uint32_t masterThreadIdx      = threadsPerQuery - 1;
  const uint32_t threadColumnEntries  = GPU_BPM_ALIGN_PEQ_ENTRY_LENGTH / GPU_UINT32_LENGTH;
  const uint32_t offsetQueryThreadIdx = gpu_get_lane_idx() - intraQueryThreadIdx;
  // Initializing back-trace data variables
  ulong2   infoCandidatePlain = GPU_TEXT_INIT, infoCandidateMasked = GPU_TEXT_INIT, infoQuery = GPU_TEXT_INIT;
  int32_t  x = minColumn, y = sizeQuery - 1;
  // Decomposing the CIGAR LUT
  const char4* const globalCigarTableLeft  = (char4*) gpu_bmp_align_cigar_lut;
  const char4* const globalCigarTableRight = (char4*) (gpu_bmp_align_cigar_lut + GPU_BMP_ALIGN_CIGAR_LUT_OFFSET);
  // Initialization for CIGAR back-trace iteration variables
  gpu_char4_t cigarOP  = GPU_CIGAR_INIT;
  gpu_bpm_align_cigar_event_t accEvent = GPU_CIGAR_NULL, event = GPU_CIGAR_NULL;
  uint32_t cigarLenght = 0, accNum = 0;
  // Performing the back-trace to extract the cigar string
  while ((y >= 0) && (x >= 0)){
    // Thread managing
    const uint32_t dpActiveThread     =  y / GPU_BPM_ALIGN_PEQ_ENTRY_LENGTH;
    const uint32_t dpLocalThreadEntry = (y % GPU_BPM_ALIGN_PEQ_ENTRY_LENGTH) / GPU_UINT32_LENGTH;
    if(intraQueryThreadIdx == dpActiveThread){
      // Read query and candidate bases
      const uint8_t encBaseRefPlain  = gpu_text_lookup(referencePlain, posCandidate + x, &infoCandidatePlain, GPU_REFERENCE_PLAIN__CHAR_LENGTH);
      const uint8_t encBaseRefMasked = gpu_text_lookup(referenceMasked, posCandidate + x, &infoCandidateMasked, GPU_REFERENCE_MASKED__CHAR_LENGTH);
      const uint8_t encBaseCandidate = (encBaseRefMasked << GPU_REFERENCE_PLAIN__CHAR_LENGTH) | encBaseRefPlain;
      const uint8_t encBaseQuery     = gpu_text_lookup(query, y, &infoQuery, GPU_BMP_ALIGN_BASE_QUERY_LENGTH);
      // Indexation for the dpMatrix element
      const uint32_t idBMP   = ((x + 1) * threadColumnEntries) + dpLocalThreadEntry;
      const uint32_t maskBMP = GPU_UINT32_ONE_MASK << (y % GPU_UINT32_LENGTH);
      // Select CIGAR operation on LUT
      const uint32_t deletion  = ((dpPV[idBMP] & maskBMP) != 0) << 2;
      const uint32_t insertion = ((dpMV[(idBMP - threadColumnEntries)] & maskBMP) != 0) << 1;
      const uint32_t match = (encBaseCandidate == encBaseQuery) && (encBaseQuery != GPU_ENC_DNA_CHAR_N);
      const uint32_t cigarEventEntry = deletion | insertion | match;
      // Recover CIGAR from DP matrix
      const char4* const localCigarTable = (leftGapAlign) ? globalCigarTableLeft : globalCigarTableRight;
      cigarOP.v4 = LDG(&localCigarTable[cigarEventEntry]);
    }
    // Communicate OP variable to the rest of the group threads
    cigarOP.s = shfl_32(cigarOP.s, offsetQueryThreadIdx + dpActiveThread);
    x += cigarOP.v4.x; y += cigarOP.v4.y; event = cigarOP.v4.z;
    // Save CIGAR string from end to start position & Resetting the CIGAR stats
    if((accEvent == GPU_CIGAR_MISSMATCH) || ((event != accEvent) && (accEvent != GPU_CIGAR_NULL))){
      if (intraQueryThreadIdx == masterThreadIdx){
      	dpCIGAR[sizeQuery - cigarLenght].event       = accEvent;
      	dpCIGAR[sizeQuery - cigarLenght].occurrences = accNum;
      }
      accNum = 0; cigarLenght++;
    }
    accEvent = event; accNum++;
  }
  // Master thread saves the last part of the cigar
  if (intraQueryThreadIdx  == masterThreadIdx){
    const gpu_bpm_align_coord_t initCood =  {(uint32_t)(x + 1), (uint32_t)(y + 1)};
    // Saving the last CIGAR status event
    dpCIGAR[sizeQuery - cigarLenght].event       = event;
    dpCIGAR[sizeQuery - cigarLenght].occurrences = accNum;
    cigarLenght++;
    // Saving the remainder semi-global deletion events
    if(y >= 0){
      const uint32_t numEvents = y + 1;
      dpCIGAR[sizeQuery - cigarLenght].event       = GPU_CIGAR_DELETION;
      dpCIGAR[sizeQuery - cigarLenght].occurrences = numEvents;
      cigarLenght++;
    }
    // Returning back-trace results
    (* initCoodRes)       = initCood;
    (* cigarLenghtRes)    = cigarLenght;
  }
}


GPU_INLINE __device__ void gpu_bpm_align_dp_matrix(uint4* const dpPV, uint4* const dpMV, const gpu_bpm_align_device_qry_entry_t* const PEQs,
												   const uint64_t* const referencePlain, const uint64_t* const referenceMasked, const uint64_t sizeReference,
												   const uint32_t sizeQuery, const uint32_t sizeCandidate, const uint64_t posCandidate,
												   const uint32_t intraQueryThreadIdx, const uint32_t threadsPerQuery,
												   uint32_t* const endMinColumn, uint32_t* const endMinScore)
{
  const uint32_t BMPS_SIZE       = GPU_BPM_ALIGN_PEQ_LENGTH_PER_CUDA_THREAD;
  const uint32_t BMPS_PER_THREAD = BMPS_SIZE / GPU_UINT32_LENGTH;

  const uint32_t laneIdx   = gpu_get_lane_idx();
  const int32_t  indexWord = ((sizeQuery - 1) % BMPS_SIZE) / GPU_UINT32_LENGTH;
  const uint32_t mask      = ((sizeQuery % GPU_UINT32_LENGTH) == 0) ? GPU_UINT32_MASK_ONE_HIGH : 1 << ((sizeQuery % GPU_UINT32_LENGTH) - 1);

  ulong2   infoCandidatePlain  = GPU_TEXT_INIT, infoCandidateMasked = GPU_TEXT_INIT;
  int32_t  score = sizeQuery, minScore = sizeQuery;
  uint32_t idColumn = 0, minColumn = 0;

  if(GPU_BPM_ALIGN_MAX_SIZE_CANDIDATE > sizeCandidate){

    uint32_t Ph[BMPS_PER_THREAD], Mh[BMPS_PER_THREAD],  Pv[BMPS_PER_THREAD], Mv[BMPS_PER_THREAD];
    uint32_t Xv[BMPS_PER_THREAD], Xh[BMPS_PER_THREAD], tEq[BMPS_PER_THREAD], Eq[BMPS_PER_THREAD];
    uint32_t sum[BMPS_PER_THREAD];

    #pragma unroll
    for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP){
      Pv[idBMP] = GPU_UINT32_ONES;
      Mv[idBMP] = 0;
    }

    dpPV[0] = gpu_compose_uintv4(Pv);
    dpMV[0] = gpu_compose_uintv4(Mv);

    for(idColumn = 0; idColumn < sizeCandidate; idColumn++){
      uint32_t PH, MH;
      const uint8_t encBasePlain  = gpu_text_lookup(referencePlain, posCandidate + idColumn, &infoCandidatePlain, GPU_REFERENCE_PLAIN__CHAR_LENGTH);
      const uint8_t encBaseMasked = gpu_text_lookup(referenceMasked, posCandidate + idColumn, &infoCandidateMasked, GPU_REFERENCE_MASKED__CHAR_LENGTH);
      const uint8_t encBase       = (encBaseMasked << GPU_REFERENCE_PLAIN__CHAR_LENGTH) | encBasePlain;
      const uint4 Eqv4 = LDG(&PEQs->bitmap[encBase]);
      gpu_decompose_uintv4(Eq, Eqv4);

      #pragma unroll
      for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP)
        Xv[idBMP] = Eq[idBMP] | Mv[idBMP];

      #pragma unroll
      for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP)
        tEq[idBMP] = Eq[idBMP] & Pv[idBMP];

      cooperative_sum(tEq, Pv, sum, intraQueryThreadIdx, BMPS_PER_THREAD);

      #pragma unroll
      for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP)
        Xh[idBMP] = (sum[idBMP] ^ Pv[idBMP]) | Eq[idBMP];

      #pragma unroll
      for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP)
        Ph[idBMP] = Mv[idBMP] | ~(Xh[idBMP] | Pv[idBMP]);

      #pragma unroll
      for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP)
        Mh[idBMP] = Pv[idBMP] & Xh[idBMP];

      PH = gpu_extract_uintv4(indexWord, Ph);
      MH = gpu_extract_uintv4(indexWord, Mh);
      score += (((PH & mask) != 0) - ((MH & mask) != 0));

      cooperative_shift(Ph, 1, intraQueryThreadIdx, BMPS_PER_THREAD);
      cooperative_shift(Mh, 1, intraQueryThreadIdx, BMPS_PER_THREAD);

      #pragma unroll
      for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP)
        Pv[idBMP] = Mh[idBMP] | ~(Xv[idBMP] | Ph[idBMP]);

      #pragma unroll
      for(uint32_t idBMP = 0; idBMP < BMPS_PER_THREAD; ++idBMP)
        Mv[idBMP] = Ph[idBMP] & Xv[idBMP];

      dpPV[idColumn + 1] = gpu_compose_uintv4(Pv);
      dpMV[idColumn + 1] = gpu_compose_uintv4(Mv);

      minColumn = (score < minScore) ? idColumn : minColumn;
      minScore  = (score < minScore) ? score    : minScore;
    }
    //Communicate the score and minimum values along thread group
    (* endMinColumn) = shfl_32(minColumn, laneIdx + (threadsPerQuery - intraQueryThreadIdx - 1));
    (* endMinScore)  = shfl_32(minScore,  laneIdx + (threadsPerQuery - intraQueryThreadIdx - 1));
  }
}

GPU_INLINE __device__ void gpu_bpm_align_local_kernel(const gpu_bpm_align_qry_entry_t* const d_queries,  const gpu_bpm_align_device_qry_entry_t* const d_PEQs, const gpu_bpm_align_qry_info_t* const d_queryInfo,
                                                      const gpu_bpm_align_cand_info_t* const d_candidateInfo, const uint64_t* const referencePlain, const uint64_t* const referenceMasked, const uint64_t sizeReference,
                                                      gpu_bpm_align_cigar_entry_t * const d_cigars, gpu_bpm_align_cigar_info_t* const d_cigarInfo,
                                                      const uint32_t idCandidate, const uint32_t intraQueryThreadIdx, const uint32_t threadsPerQuery)
{
  const uint32_t masterThreadIdx                      = threadsPerQuery - 1;
  const uint32_t sizeCandidate                        = d_candidateInfo[idCandidate].size;
  const uint64_t posCandidate                         = d_candidateInfo[idCandidate].position;
  if((posCandidate + sizeCandidate) < sizeReference){
    // Data characterization
    const uint32_t idQuery                             = d_candidateInfo[idCandidate].idQuery;
    const uint32_t idCigar                             = idCandidate;
    const uint32_t sizeQuery                           = d_queryInfo[idQuery].size;
    const bool     leftGapAlign                        = d_candidateInfo[idCandidate].leftGapAlign;
    const uint32_t offsetCigarStart					           = d_cigarInfo[idCandidate].offsetCigarStart;
    // Data Buffers
    const uint64_t* const query                        = (uint64_t*) (d_queries + d_queryInfo[idQuery].posEntryBase);
    const gpu_bpm_align_device_qry_entry_t* const PEQs = d_PEQs + d_queryInfo[idQuery].posEntryPEQ + intraQueryThreadIdx;
    gpu_bpm_align_cigar_entry_t* const cigar           = d_cigars + offsetCigarStart;
    gpu_bpm_align_cigar_info_t* const cigarInfo        = d_cigarInfo + idCigar;

    // Local Memory (DP Matrix allocated in the CUDA stack)
    uint4 dpPV[GPU_BPM_ALIGN_MAX_SIZE_CANDIDATE];
    uint4 dpMV[GPU_BPM_ALIGN_MAX_SIZE_CANDIDATE];
    // DP matrix conversion for back-trace data-layout specialization
    const uint32_t* const dpPV4 = (uint32_t*) dpPV;
    const uint32_t* const dpMV4 = (uint32_t*) dpMV;

    //Return values for align DP matrix
    uint32_t minColumn = 0, minScore = sizeQuery;
    //Return values for align back-trace
    gpu_bpm_align_coord_t initCood = {0,0};
    uint32_t cigarLenght = 0;

    gpu_bpm_align_dp_matrix(dpPV,  dpMV, PEQs, referencePlain, referenceMasked, sizeReference, sizeQuery, sizeCandidate, posCandidate,
		                    intraQueryThreadIdx, threadsPerQuery, &minColumn, &minScore);
    gpu_bpm_align_backtrace(dpPV4, dpMV4, query, referencePlain, referenceMasked, sizeReference, posCandidate,
		                    cigar, leftGapAlign, minColumn, sizeQuery,
		                    intraQueryThreadIdx, threadsPerQuery, &initCood, &cigarLenght);

    // Return the cigar results
    if (intraQueryThreadIdx  == masterThreadIdx){
      cigarInfo->initCood       = initCood;
      cigarInfo->endCood.x      = minColumn;
      cigarInfo->endCood.y      = sizeQuery - 1;
      cigarInfo->cigarStartPos  = offsetCigarStart + sizeQuery - cigarLenght + 1;
      cigarInfo->cigarLenght    = cigarLenght;
    }
  }
}

__global__ void gpu_bpm_align_kernel(const gpu_bpm_align_qry_entry_t* const d_queries,  const gpu_bpm_align_device_qry_entry_t * const d_PEQs, const gpu_bpm_align_qry_info_t* const d_queryInfo,
                                     const gpu_bpm_align_cand_info_t* const d_candidateInfo, const uint32_t* const d_reorderBuffer,
                                     const uint64_t* const d_referencePlain, const uint64_t* const d_referenceMasked, const uint64_t referenceSize,
                                     gpu_bpm_align_cigar_entry_t * const d_cigars, gpu_bpm_align_cigar_info_t* const d_cigarInfo, const uint32_t numCigars,
                                     const uint32_t* const d_initPosPerBucket, const uint32_t* const d_initWarpPerBucket, const uint32_t* const d_endPosPerBucket, const bool updateScheduling)
{
  // Thread Identification
  const uint32_t globalThreadIdx = gpu_get_thread_idx();
  uint32_t intraQueryThreadIdx = 0, threadsPerQuery = 0;
  // Identification tracking of the candidate task ID
  uint32_t idCandidate;
  // Rescheduling thread mapping and thread set distribution
  gpu_scheduler_scatter_work(globalThreadIdx, d_initWarpPerBucket, d_initPosPerBucket, d_endPosPerBucket,
                             d_reorderBuffer, updateScheduling,
                             &idCandidate, &intraQueryThreadIdx, &threadsPerQuery);
  // Call to the device align BPM process for the active threads
  if ((idCandidate < numCigars) && (idCandidate != GPU_SCHEDULER_DISABLED_TASK)){
    // Update the buffer input/output for the thread re-scheduling
    gpu_bpm_align_local_kernel(d_queries, d_PEQs, d_queryInfo, d_candidateInfo,
    		                   d_referencePlain, d_referenceMasked, referenceSize,
    		                   d_cigars, d_cigarInfo,
                               idCandidate, intraQueryThreadIdx, threadsPerQuery);
  }
}

extern "C"
gpu_error_t gpu_bpm_align_process_buffer(gpu_buffer_t *mBuff)
{
  // Internal buffer handles
  const gpu_reference_buffer_t* const             ref               =  mBuff->reference;
  const gpu_bpm_align_queries_buffer_t* const     qry               = &mBuff->data.abpm.queries;
  const gpu_bpm_align_candidates_buffer_t* const  cand              = &mBuff->data.abpm.candidates;
  const gpu_scheduler_buffer_t* const             rebuff            = &mBuff->data.abpm.reorderBuffer;
  const gpu_bpm_align_cigars_buffer_t* const      cigar             = &mBuff->data.abpm.cigars;
  // Device properties
  const hipStream_t                              idStream          =  mBuff->listStreams[mBuff->idStream];
  const uint32_t                                  idSupDev          =  mBuff->idSupportedDevice;
  const gpu_device_info_t* const                  device            =  mBuff->device[idSupDev];
  // Buffer size parameters information
  const uint32_t                                  numQueries        =  qry->numQueries;
  const uint32_t                                  numQueryBases     =  qry->totalQueriesBases;
  const uint32_t                                  numQueryPEQs      =  qry->totalQueriesPEQs;
  const uint32_t                                  numCandidates     =  cand->numCandidates;
  // Buffer size parameters for maximal threshold
  const uint32_t                                  maxQueries        =  mBuff->data.abpm.maxQueries;
  const uint32_t                                  maxQueryBases     =  mBuff->data.abpm.maxQueryBases;
  const uint32_t                                  maxQueryPEQs      =  mBuff->data.abpm.maxPEQEntries;
  const uint32_t                                  maxCandidates     =  mBuff->data.abpm.maxCandidates;
  const uint32_t                                  maxCigars         =  mBuff->data.abpm.maxCigars;
  // Cigar results information
  const uint32_t                                  numCigars         = cigar->numCigars;
  gpu_bpm_align_cigar_info_t*                     cigarsInfo        = cigar->d_cigarsInfo;
  // Thread work distribution
  dim3 blocksPerGrid, threadsPerBlock;
  const uint32_t numThreads = rebuff->numWarps * GPU_WARP_SIZE;
  gpu_device_kernel_thread_configuration(device, numThreads, &blocksPerGrid, &threadsPerBlock);
  // Sanity-check (checks buffer overflowing)
  if((numQueries > maxQueries) || (numCandidates > maxCandidates) || (numQueryPEQs > maxQueryPEQs) ||
     (numQueryBases > maxQueryBases) || (numCigars > maxCigars))
    return(E_OVERFLOWING_BUFFER);
  // Launching the BPM align kernel on device
  gpu_bpm_align_kernel<<<blocksPerGrid, threadsPerBlock, 0, idStream>>>(qry->d_queries, (gpu_bpm_align_device_qry_entry_t *) qry->d_peq, qry->d_qinfo,
                                                                        cand->d_candidatesInfo, rebuff->threadMapScheduler.d_reorderBuffer,
                                                                        ref->d_reference_plain[idSupDev], ref->d_reference_masked[idSupDev], ref->size,
                                                                        cigar->d_cigars, cigarsInfo, numCigars,
                                                                        rebuff->d_initPosPerBucket, rebuff->d_initWarpPerBucket, rebuff->d_endPosPerBucket,
                                                                        mBuff->data.abpm.queryBinning);
  return(SUCCESS);
}

#endif /* GPU_BPM_ALIGN_CU_ */
